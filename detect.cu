#include "hip/hip_runtime.h"
/**
 * Detect.cu
 * Provides functions for lane detection
 * 
 * @author Oscar Morales Ponce
 * @author Jared Coleman
 * @version 1.0 12/02/17
 */

using namespace std;

#include <fstream>
#include <string>
#include <cmath>

#include "opencv2/opencv.hpp"
#include "opencv2/gpu/gpu.hpp"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

#include "polifitgsl.h"

using namespace cv;

/**
 * Class implements a lane
 */
class Lane
{
public:
	int degree; //degree of polynomial that defines lanes
	vector<double> l_params; //array of size degree. Defines coefficients for left lane curve.
	vector<double> r_params; //array of size degree. Defines coefficients for right lane curve.
	double filter; //filter for curve to remove jitter. lane = old_lane*filter + new_lane*(1-filter).
	
	/**
	 * Only provided constructor for Lane.
	 * @param degree Degree of polynomial that defines lanes.
	 * @param filter Filter for curve to remove jitter.
	 */ 
	Lane(int degree, double filter = 0.9) 
		: degree(degree)
		, l_params(vector<double>(degree, (double)nan("1")))
		, r_params(vector<double>(degree, (double)nan("1")))
		, filter(filter)
	{ 
		if (this->filter < 0.0 || this->filter > 1.0) this->filter = 0.9;
	}
	
	/**
	 * Updates left and right lane polynomial coefficients.
	 * @param l_new Array of size degree. Defines coefficients for left lane curve.
	 * @param r_new Array of size degree. Defines coefficients for right lane curve.
	 */
	void update(vector<double> l_new, vector<double> r_new)
	{
		if (l_params[0] != l_params[0])
		{
			l_params = l_new;
			r_params = r_new;
		}
		else
		{
			for (int i = 0; i < degree; i++)
			{
				l_params[i] = filter * l_params[i] + (1 - filter) * l_new[i];
				r_params[i] = filter * r_params[i] + (1 - filter) * r_new[i];
			}
		}
	}
};

/**
 * Defines configuration parameter provided by config.txt
 */
struct Config
{
	string video_file; // video file to read
	int lane_degree; // degree of polynomial that defines lanes
	double lane_filter; // filter for curve to remove jitter. lane = old_lane*filter + new_lane*(1-filter).
	int lane_start_threshold; // +/- pixel threshold to look for lane. 
	int left_lane_start; // percentage of width to start looking for left lane
	int right_lane_start; // percentage of width to start looking for right lane
	int row_step; // stride for stepping through image rows
	int col_step; // stride for stepping through image columns
};

//Configuration
Config config;

/**
 * Reads configuration file and sets configuration parameters.
 * @return Data structure of configuration parameters
 */
Config getConfig()
{
	Config config;
	
	ifstream ifs("config.txt");
	istringstream is_file(string((std::istreambuf_iterator<char>(ifs)),
                 std::istreambuf_iterator<char>()));

	string line;
	while( getline(is_file, line) )
	{
	  istringstream is_line(line);
	  string key;
	  if( getline(is_line, key, '=') )
	  {
		string value;
		if( getline(is_line, value) ) 
		{
			if (key =="video_file") config.video_file = string(value); 
			else if (key == "lane_degree") config.lane_degree = stoi(value);
			else if (key == "lane_filter") config.lane_filter = stod(value);
			else if (key == "lane_start_threshold") config.lane_start_threshold = stoi(value);
			else if (key == "left_lane_start") config.left_lane_start = stoi(value);
			else if (key == "right_lane_start") config.right_lane_start = stoi(value);
			else if (key == "row_step") config.row_step = stoi(value);
			else if (key == "col_step") config.col_step = stoi(value);
		}
	  }
	}
	return config;
}

/**
 * Thresholds the image. Uses GPU acceleration.
 * Process:
 *   1. Convert image to grayscale
 *   2. Blur the image to remove noise (gaussian)
 *   3. threshold image (binary)
 * @param img Image to threshold
 */
void thresh(Mat &img)
{
	gpu::GpuMat g1;
	gpu::GpuMat g2;
	
	g1.upload(img);
	gpu::cvtColor(g1, g2, CV_BGR2GRAY);
	
	gpu::GaussianBlur(g2, g2, Size( 7, 7 ), 1.5, 1.5 );
	gpu::threshold(g2, g2, 185, 255, THRESH_BINARY);
	g2.download(img);
}

/**
 * Converts an image to/from birdseye view)
 * @param img Image to convert
 * @param undo If true, convert to birdseye. If false, convert from birdseye.
 */
void birdseye(Mat &img, bool undo=false)
{
	int width = img.cols;
	int height = img.rows;
	vector<Point2f> src = {Point2f(width*0.44,height*0.20), Point2f(width*0.56,height*0.20), Point2f(width*1.00,height*0.85), Point2f(width*0.00,height*0.85)};
	vector<Point2f> dst = {Point2f(width*0.20,height*0.00), Point2f(width*0.80,height*0.00), Point2f(width*0.80,height*1.00), Point2f(width*0.20,height*1.00)};
	
	Mat m;
	if (undo) m = getPerspectiveTransform(&dst[0], &src[0]);
	else m = getPerspectiveTransform(&src[0], &dst[0]);
	
	gpu::GpuMat g1(img);
	gpu::GpuMat g2;
	warpPerspective(g1, g2, m, Size(width, height));
	g2.download(img);
}	

/**
 * Evaluates a polynomial expression.
 * @param params Array of polynomial coefficients.
 * @param degree Degree of polynomial (size of params).
 * @param x Polynomial input.
 * @return Evaluated expression.
 */
int polynomial(const double *params, int degree, double x)
{
	double val = 0;
	for (int i = 0; i < degree; i++)
	{
		val += params[i] * pow(x, i);
	}
	return (int)val;
}	

/**
 * Get lanes
 * @param img Frame from video.
 * @param lane Detected lane.
 */
void getLanes(const Mat &img, Lane &lane)
{
	static int row_step = config.row_step;
	static int col_step = config.col_step;
	static int d = config.lane_start_threshold;
	
	Mat img_thresh = img.clone();
	thresh(img_thresh);
	birdseye(img_thresh);
	int width = img_thresh.cols;
	int height = img_thresh.rows;
	
	int left = width * config.left_lane_start / 100;
	int right = width * config.right_lane_start / 100;
	
	vector<double> lx;
	vector<double> rx;
	vector<double> ly;
	vector<double> ry;
	
	//Loop through frame rows at row_step
	for (int i = height-1; i >= 0; i-=row_step)
	{
		//Loop through left side
		lx.push_back(left);
		ly.push_back(i);
		for (int j = left + d; j >= left - d; j-=col_step)
		{
			if (img_thresh.at<uchar>(i, j) == 255) 
			{
				lx.back() = j;
				left = j;
				break;
			}
		}
		
		//Loop through right side
		rx.push_back(right);
		ry.push_back(i);
		for (int j = right - d; j < right + d; j+=col_step)
		{
			if (img_thresh.at<uchar>(i, j) == 255) 
			{
				rx.back() = j;
				right = j;
				break;
			}
		}
		 
	}
	
	vector<double> l_new(lane.degree, 0.0);
	vector<double> r_new(lane.degree, 0.0);
	polynomialfit(lx.size(), lane.degree, &ly[0], &lx[0], &l_new[0]);
	polynomialfit(rx.size(), lane.degree, &ry[0], &rx[0], &r_new[0]);
	
	lane.update(l_new, r_new);
}

/**
 * Draws lane on an image.
 * @param img Image on which to draw lane.
 * @param lane Lane to draw.
 */
void drawLane(Mat &img, const Lane &lane)
{
	Mat blank(img.size(), img.type(), Scalar(0, 0, 0));
	for (int i = 0; i < img.rows; i++)
	{
		circle(blank, Point(polynomial(&(lane.l_params)[0], lane.degree, i), i), 3, Scalar(150, 0, 0), 3);
		circle(blank, Point(polynomial(&(lane.r_params)[0], lane.degree, i), i), 3, Scalar(150, 0, 0), 3); 
	}
	birdseye(blank, true);
	for (int i = 0; i < img.rows; i+=2)
	{
		for (int j = 0; j < img.cols; j+=2)
		{
			if (blank.at<Vec3b>(i, j)[0] == 150)
				circle(img, Point(j, i), 1, Scalar(150, 0, 0), 1);
		}
	}
}

int main(int argc, char* argv[])
{
	config = getConfig();
    Lane lane(config.lane_degree, config.lane_filter);
		
	VideoCapture cap(config.video_file); 
    if(!cap.isOpened()) return -1;
    
    namedWindow("output", 1);
    Mat frame;
    while(true)
    {
		cap >> frame;
		//-------------------------------------------------------//
		getLanes(frame, lane);
		drawLane(frame, lane);
		
		//-------------------------------------------------------//
		imshow("output", frame);
		if(waitKey(1) >= 0) break;
	}
}
