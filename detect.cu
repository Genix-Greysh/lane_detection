#include "hip/hip_runtime.h"
using namespace std;

#include <fstream>
#include <string>

#include "opencv2/opencv.hpp"
//#include "opencv2/core/cuda.hpp"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

#include "polifitgsl.h"

using namespace cv;

class Lane
{
public:
	int degree;
	double *l_params;
	double *r_params;
	
	Lane(int d) 
	{ 
		degree = d; 
		l_params = new double[d];
		r_params = new double[d];
	}
	
	~Lane()
	{
		delete[] l_params;
		delete[] r_params;
	}
};

struct Config
{
	string video_file;
	int lane_degree;
	int lane_start_threshold;
	int left_lane_start;
	int right_lane_start;
	int row_step;
	int col_step;
};

Config config;

Config getConfig()
{
	Config config;
	
	ifstream ifs("config.txt");
	istringstream is_file(string((std::istreambuf_iterator<char>(ifs)),
                 std::istreambuf_iterator<char>()));

	string line;
	while( getline(is_file, line) )
	{
	  istringstream is_line(line);
	  string key;
	  if( getline(is_line, key, '=') )
	  {
		string value;
		if( getline(is_line, value) ) 
		{
			if (key =="video_file") config.video_file = string(value); 
			else if (key == "lane_degree") config.lane_degree = stoi(value);
			else if (key == "lane_start_threshold") config.lane_start_threshold = stoi(value);
			else if (key == "left_lane_start") config.left_lane_start = stoi(value);
			else if (key == "right_lane_start") config.right_lane_start = stoi(value);
			else if (key == "row_step") config.row_step = stoi(value);
			else if (key == "col_step") config.col_step = stoi(value);
		}
	  }
	}
	return config;
}

void thresh(Mat &img)
{
	cvtColor(img, img, CV_BGR2GRAY);
	GaussianBlur( img, img, Size( 7, 7 ), 1.5, 1.5 );
	threshold(img, img, 185, 255, THRESH_BINARY);
}

void birdseye(Mat &img, bool undo=false)
{
	int width = img.cols;
	int height = img.rows;
	vector<Point2f> src = {Point2f(width*0.44,height*0.20), Point2f(width*0.56,height*0.20), Point2f(width*1.00,height*0.85), Point2f(width*0.00,height*0.85)};
	vector<Point2f> dst = {Point2f(width*0.20,height*0.00), Point2f(width*0.80,height*0.00), Point2f(width*0.80,height*1.00), Point2f(width*0.20,height*1.00)};
	
	Mat m;
	if (undo) m = getPerspectiveTransform(&dst[0], &src[0]);
	else m = getPerspectiveTransform(&src[0], &dst[0]);
	warpPerspective(img, img, m, Size(width, height));
}	

int polynomial(double *params, int degree, double x)
{
	double val = 0;
	for (int i = 0; i < degree; i++)
	{
		val += params[i] * pow(x, i);
	}
	return (int)val;
}	

Lane getLanes(const Mat &img)
{
	static int row_step = config.row_step;
	static int col_step = config.col_step;
	static int d = config.lane_start_threshold;
	
	Mat img_thresh = img.clone();
	thresh(img_thresh);
	birdseye(img_thresh);
	int width = img_thresh.cols;
	int height = img_thresh.rows;
	
	int left = width * config.left_lane_start / 100;
	int right = width * config.right_lane_start / 100;
	
	Lane lane(config.lane_degree);
	thrust::device_vector<double> lx;
	thrust::device_vector<double> rx;
	thrust::device_vector<double> ly;
	thrust::device_vector<double> ry;
	
	//Loop through frame rows
	for (int i = height-1; i >= 0; i-=row_step)
	{
		lx.push_back(left);
		ly.push_back(i);
		for (int j = left + d; j >= left - d; j-=col_step)
		{
			if (img_thresh.at<uchar>(i, j) == 255) 
			{
				lx.back() = j;
				left = j;
				break;
			}
		}
		
		rx.push_back(right);
		ry.push_back(i);
		for (int j = right - d; j < right + d; j+=col_step)
		{
			if (img_thresh.at<uchar>(i, j) == 255) 
			{
				rx.back() = j;
				right = j;
				break;
			}
		}
		 
	}
	
	polynomialfit(lx.size(), lane.degree, &ly[0], &lx[0], lane.l_params);
	polynomialfit(rx.size(), lane.degree, &ry[0], &rx[0], lane.r_params);
	
	return lane;
}

void drawLane(Mat &img, const Lane &lane)
{
	//draw
	Mat blank(img.size(), img.type(), Vec3b(0, 0, 0));
	for (int i = 0; i < img.rows; i++)
	{
		circle(blank, Point(polynomial(lane.l_params, lane.degree, i), i), 3, Scalar(150, 0, 0), 3);
		circle(blank, Point(polynomial(lane.r_params, lane.degree, i), i), 3, Scalar(150, 0, 0), 3); 
	}
	birdseye(blank, true);
	for (int i = 0; i < img.rows; i++)
	{
		for (int j = 0; j < img.cols; j++)
		{
			if (blank.at<Vec3b>(i, j)[0] == 150)
				circle(img, Point(j, i), 1, Scalar(150, 0, 0), 1);
		}
	}
}

int main(int argc, char* argv[])
{
	config = getConfig();
		
	VideoCapture cap(config.video_file); 
    if(!cap.isOpened()) return -1;
    
    namedWindow("output", 1);
    Mat frame;
    while(true)
    {
		cap >> frame;
		//-------------------------------------------------------//
		Lane lane = getLanes(frame);
		//drawLane(frame, lane);
		
		//-------------------------------------------------------//
		imshow("output", frame);
		if(waitKey(1) >= 0) break;
	}
}
